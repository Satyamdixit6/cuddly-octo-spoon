#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void vector_add_kernel(
    const scalar_t* __restrict__ a,
    const scalar_t* __restrict__ b,
    scalar_t* __restrict__ c,
    int size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        c[index] = a[index] + b[index];
    }
}

at::Tensor vector_add_forward_cuda(at::Tensor a, at::Tensor b) {
    auto c = torch::empty_like(a);
    int size = a.numel();

    const int threads = 1024;
    const int blocks = (size + threads - 1) / threads;

    AT_DISPATCH_FLOATING_TYPES(a.scalar_type(), "vector_add_kernel", ([&] {
        vector_add_kernel<scalar_t><<<blocks, threads>>>(
            a.data_ptr<scalar_t>(),
            b.data_ptr<scalar_t>(),
            c.data_ptr<scalar_t>(),
            size);
    }));

    hipError_t err = hipGetLastError();
    TORCH_CHECK(err == hipSuccess, "CUDA kernel failed : ", hipGetErrorString(err));

    return c;
}