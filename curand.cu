#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void generateRandomKernel(float* output, int n, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        hiprandState_t state;
        hiprand_init(seed, idx, 0, &state);
        output[idx] = hiprand_uniform(&state);
    }
}