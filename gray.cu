#include <hip/hip_runtime.h>

__global__ void grayscaleKernel(const uchar3* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * width + x;

    if (x < width && y < height) {
        uchar3 pixel = input[idx];
        output[idx] = (unsigned char)(0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z);
    }
}