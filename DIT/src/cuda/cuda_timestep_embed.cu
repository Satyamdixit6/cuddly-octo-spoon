#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_CUDA_ERROR(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s:%d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        TORCH_CHECK(false, "CUDA error"); \
    } \
} while(0)

#define TILE_DIM 16

__global__ void timestep_embed_kernel(const float* __restrict__ t,
                                     const float* __restrict__ freqs,
                                     float* __restrict__ embed,
                                     int N, int dim, int half) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int d = blockIdx.y * blockDim.y + threadIdx.y;
    if (n < N && d < dim) {
        __shared__ float freqs_shared[256];
        for (int i = threadIdx.y; i < half; i += blockDim.y) {
            freqs_shared[i] = freqs[i];
        }
        __syncthreads();

        float t_val = t[n];
        int index = n * dim + d;
        if (d < half) {
            embed[index] = cosf(t_val * freqs_shared[d]);
        } else {
            embed[index] = sinf(t_val * freqs_shared[d - half]);
        }
    }
}

__global__ void linear_kernel(const float* __restrict__ A,
                             const float* __restrict__ B,
                             const float* __restrict__ bias,
                             float* __restrict__ out,
                             int N, int K, int M) {
    int row = blockIdx.y * TILE_DIM + threadIdx.y;
    int col = blockIdx.x * TILE_DIM + threadIdx.x;
    float sum = 0.0f;
    __shared__ float sA[TILE_DIM][TILE_DIM];
    __shared__ float sB[TILE_DIM][TILE_DIM];

    for (int t = 0; t < (K + TILE_DIM - 1) / TILE_DIM; t++) {
        int A_col = t * TILE_DIM + threadIdx.x;
        if (row < N && A_col < K)
            sA[threadIdx.y][threadIdx.x] = A[row * K + A_col];
        else
            sA[threadIdx.y][threadIdx.x] = 0.0f;

        int B_row = t * TILE_DIM + threadIdx.y;
        if (col < M && B_row < K)
            sB[threadIdx.y][threadIdx.x] = B[col * K + B_row];
        else
            sB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int i = 0; i < TILE_DIM; i++) {
            sum += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < N && col < M) {
        out[row * M + col] = sum + bias[col];
    }
}

__global__ void linear_silu_kernel(const float* __restrict__ A,
                                  const float* __restrict__ B,
                                  const float* __restrict__ bias,
                                  float* __restrict__ out,
                                  int N, int K, int M) {
    int row = blockIdx.y * TILE_DIM + threadIdx.y;
    int col = blockIdx.x * TILE_DIM + threadIdx.x;
    float sum = 0.0f;
    __shared__ float sA[TILE_DIM][TILE_DIM];
    __shared__ float sB[TILE_DIM][TILE_DIM];

    for (int t = 0; t < (K + TILE_DIM - 1) / TILE_DIM; t++) {
        int A_col = t * TILE_DIM + threadIdx.x;
        if (row < N && A_col < K)
            sA[threadIdx.y][threadIdx.x] = A[row * K + A_col];
        else
            sA[threadIdx.y][threadIdx.x] = 0.0f;

        int B_row = t * TILE_DIM + threadIdx.y;
        if (col < M && B_row < K)
            sB[threadIdx.y][threadIdx.x] = B[col * K + B_row];
        else
            sB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int i = 0; i < TILE_DIM; i++) {
            sum += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < N && col < M) {
        float x = sum + bias[col];
        float sig = 1.f / (1.f + expf(-x));
        out[row * M + col] = x * sig;
    }
}

torch::Tensor timestep_embedding_forward(torch::Tensor t, torch::Tensor freqs, int dim) {
    CHECK_CUDA(t); CHECK_CUDA(freqs);
    CHECK_CONTIGUOUS(t); CHECK_CONTIGUOUS(freqs);

    int N = t.size(0);
    int half = dim / 2;
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
    auto embed = torch::empty({N, dim}, options);
    dim3 blockDim(16, 16);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (dim + blockDim.y - 1) / blockDim.y);
    timestep_embed_kernel<<<gridDim, blockDim>>>(
        t.data_ptr<float>(), freqs.data_ptr<float>(), embed.data_ptr<float>(), N, dim, half
    );
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    return embed;
}

torch::Tensor linear_forward(torch::Tensor A, torch::Tensor weight, torch::Tensor bias) {
    CHECK_CUDA(A); CHECK_CUDA(weight); CHECK_CUDA(bias);
    CHECK_CONTIGUOUS(A); CHECK_CONTIGUOUS(weight); CHECK_CONTIGUOUS(bias);

    int N = A.size(0);
    int K = A.size(1);
    int M = weight.size(0);
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
    auto out = torch::empty({N, M}, options);
    dim3 blockDim(TILE_DIM, TILE_DIM);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);
    linear_kernel<<<gridDim, blockDim>>>(
        A.data_ptr<float>(), weight.data_ptr<float>(), bias.data_ptr<float>(), out.data_ptr<float>(), N, K, M
    );
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    return out;
}

torch::Tensor linear_silu_forward(torch::Tensor A, torch::Tensor weight, torch::Tensor bias) {
    CHECK_CUDA(A); CHECK_CUDA(weight); CHECK_CUDA(bias);
    CHECK_CONTIGUOUS(A); CHECK_CONTIGUOUS(weight); CHECK_CONTIGUOUS(bias);

    int N = A.size(0);
    int K = A.size(1);
    int M = weight.size(0);
    auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
    auto out = torch::empty({N, M}, options);
    dim3 blockDim(TILE_DIM, TILE_DIM);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x, (N + blockDim.y - 1) / blockDim.y);
    linear_silu_kernel<<<gridDim, blockDim>>>(
        A.data_ptr<float>(), weight.data_ptr<float>(), bias.data_ptr<float>(), out.data_ptr<float>(), N, K, M
    );
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    return out;
}

torch::Tensor timestep_embed_mlp_forward(torch::Tensor t_freq,
                                        torch::Tensor weight1, torch::Tensor bias1,
                                        torch::Tensor weight2, torch::Tensor bias2) {
    auto hidden = linear_silu_forward(t_freq, weight1, bias1);
    auto out = linear_forward(hidden, weight2, bias2);
    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("timestep_embedding_forward", &timestep_embedding_forward, "Timestep embedding forward (CUDA)");
    m.def("linear_forward", &linear_forward, "Linear forward (CUDA)");
    m.def("linear_silu_forward", &linear_silu_forward, "Linear + SiLU forward (CUDA)");
    m.def("timestep_embed_mlp_forward", &timestep_embed_mlp_forward, "Full MLP forward (CUDA)");
}