#include <hip/hip_runtime.h>

__global__ void reduceSumSharedKernel(const float* g_idata, float* g_odata, unsigned int n) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        sdata[tid] = g_idata[i];
    } else {
        sdata[tid] = 0.0f;
    }
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}